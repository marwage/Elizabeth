#include "hip/hip_runtime.h"
// Copyright 2020 Marcel Wagenländer

#include <math.h>

#include "axpby.h"


__global__ void axpby(float alpha, float *x, float beta, float *y, int num_elements) {
    // n rows, i is blockIdx.x
    // m columns, j is threadIdx.x
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) y[idx] = (alpha * x[idx]) + (beta * y[idx]);
}

void xpy(float alpha, float *x, float beta, float *y, int num_elements) {
    int num_threads = 1024;
    int num_blocks = ceil((float) num_elements / (float) num_threads);
    axpby<<<num_blocks, num_threads>>>(alpha, x, beta, y, num_elements);
}
